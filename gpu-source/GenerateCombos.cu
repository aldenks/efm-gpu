#include "hip/hip_runtime.h"
#include "GenerateCombos.h"

__device__
void scalePathway(float* result, float* p1, float* p2, float scale, int metaboliteCount){
   for(int i = 0; i < metaboliteCount; i++){
      result[i] = p1[i] + p2[i] * scale;
   }
}

__global__
void generateCombinations(int* bins, int* indicies, int inputIndex, int numberOfBins, int metabolite, int metaboliteCount, BinaryVector *reactions, float *metaboliteCoefficients){
   int tid = blockIdx.x + blockDim.x + threadIdx.x;
   if(tid >= numberOfBins){
      return;
   }
   int writeIndex = indicies[tid];
   bins += tid;
   int writeCount = *bins;
   inputIndex = circularIndex(inputIndex + tid);
   BinaryVector inputReaction = reactions[inputIndex];
   int outputIndex;
   float met1, met2;
   for(int i = 0; i < writeCount; i++, writeIndex = circularIndex(writeIndex + 1)){
      bins += numberOfBins;
      outputIndex = *bins;
      reactions[writeIndex] = inputReaction | reactions[outputIndex];
      met1 = metaboliteCoefficients[metaboliteCount * inputIndex + metabolite];
      met2 = metaboliteCoefficients[metaboliteCount * outputIndex + metabolite];
      if(met1 < met2){
         scalePathway(metaboliteCoefficients + metaboliteCount * writeIndex, metaboliteCoefficients + metaboliteCount * inputIndex, metaboliteCoefficients + metaboliteCount * outputIndex, met1/met2, metaboliteCount);
      }else{
         scalePathway(metaboliteCoefficients + metaboliteCount * writeIndex, metaboliteCoefficients + metaboliteCount * outputIndex, metaboliteCoefficients + metaboliteCount * inputIndex, met2/met1, metaboliteCount);
      }
   }
}

int generateCombinations(int metabolite, int numberOfBins, int nextFreePathwayIndex){
   hipMemcpy(h_combinationBinCounts, d_combinationBins, numberOfBins * sizeof(int), hipMemcpyDeviceToHost);
   h_combinationWriteIndices[0] = nextFreePathwayIndex;
   int newComboCount = h_combinationBinCounts[0];
   printf("%i -- %i combos\n", 0, h_combinationBinCounts[0]);
   for(int i = 1; i < numberOfBins; i++){
      printf("%i -- %i combos\n", i, h_combinationBinCounts[i]);
      h_combinationWriteIndices[i] = h_combinationWriteIndices[i - 1] + h_combinationBinCounts[i - 1];
      newComboCount += h_combinationBinCounts[i];
   }
   hipMemcpy(d_combinationWriteIndices, h_combinationWriteIndices, numberOfBins * sizeof(int), hipMemcpyHostToDevice);
   int numBlocks = (numberOfBins / MAX_THREADS_PER_BLOCK) + 1;
   generateCombinations << < numBlocks, MAX_THREADS_PER_BLOCK >> > (d_combinationBins, d_combinationWriteIndices, pathwayStartIndex, numberOfBins, metabolite, metaboliteCount, d_binaryVectors, d_metaboliteCoefficients);
   return newComboCount;
}
