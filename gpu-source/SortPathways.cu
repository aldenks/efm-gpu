#include "hip/hip_runtime.h"
#include "SortPathways.h"

__global__
void sortInputPathways(BinaryVector *reactions, float *metaboliteCoefficients,
		  int startIndex, int numToSort, int numReactions, 
		  int numberOfMetabolites, int metaboliteToRemove){

  int tid = blockIdx.x + blockDim.x + threadIdx.x;

  if ( tid == 0 ) {
    int start = startIndex;
    int end = startIndex + numToSort;

    //While the pointers do not overlap
    while(start < end){
      bool is_input_1 = metaboliteCoefficients[start * numberOfMetabolites + 
					       metaboliteToRemove] < NEG_ZERO;
      bool is_input_2 = metaboliteCoefficients[end * numberOfMetabolites + 
					       metaboliteToRemove] < NEG_ZERO;
      if(is_input_1) {
	//Skip this one
	start++;
      } else if(is_input_2){
	//swap the two reactions
	BinaryVector temp = reactions[end];
	reactions[end] = reactions[start];
	reactions[start] = temp;
	
	//swap the two metaboliteCoefficients for pathways
	float tempCoefficient;
	for(int i = 0; i < numberOfMetabolites; ++i){
	  tempCoefficient = metaboliteCoefficients[end * numberOfMetabolites + i];
	  metaboliteCoefficients[end * numberOfMetabolites + i] = 
	    metaboliteCoefficients[start * numberOfMetabolites + i];
	  metaboliteCoefficients[start * numberOfMetabolites + i] = tempCoefficient;
	}
	//move forward
	start++;
	end++;
      } else {
	//Not an input, don't care
	end++;
      }
    }
  }
}

__global__
void sortOutputPathways(BinaryVector *reactions, float *metaboliteCoefficients,
		  int startIndex, int numToSort, int numReactions, 
		  int numberOfMetabolites, int metaboliteToRemove){

  int tid = blockIdx.x + blockDim.x + threadIdx.x;

  if ( tid == 0 ) {
    int start = startIndex;
    int end = startIndex + numToSort;

    //While the pointers do not overlap
    while(start < end){
      bool is_output_1 = metaboliteCoefficients[start * numberOfMetabolites + 
					       metaboliteToRemove] > ZERO;
      bool is_output_2 = metaboliteCoefficients[end * numberOfMetabolites + 
					       metaboliteToRemove] > ZERO;
      if(is_output_1) {
	//Skip this one
	start++;
      }	if(is_output_2){
	//swap the two reactions
	BinaryVector temp = reactions[end];
	reactions[end] = reactions[start];
	reactions[start] = temp;
	
	//swap the two metaboliteCoefficients for pathways
	float tempCoefficient;
	for(int i = 0; i < numberOfMetabolites; ++i){
	  tempCoefficient = metaboliteCoefficients[end * numberOfMetabolites + i];
	  metaboliteCoefficients[end * numberOfMetabolites + i] = 
	    metaboliteCoefficients[start * numberOfMetabolites + i];
	  metaboliteCoefficients[start * numberOfMetabolites + i] = tempCoefficient;
	}
	//move forward
	start++;
	end++;
      } else {
	//Not an output, don't care
	end++;
      }
    }
  }
}

__global__ 
void dependencyCheck(BinaryVector *reactions, int *bins, int batch_size, 
		     int num_inputs, int output_start, int non_part_start,
		     int pathwayCounts){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int count = 0;
  if(tid < num_inputs){
    BinaryVector input = reactions[tid];
    BinaryVector output, combo, pathway;
    bool is_unique_and_independent = false;
    for(int i = 0; i < batch_size; ++i){
      output = reactions[output_start + i];
      combo = input | output;
      for (int j = 0; is_unique_and_independent && j < pathwayCounts; ++j){
	if(j == tid) continue; //skip this input
	if(j == output_start + i) continue; //skip this output
	pathway = reactions[j];
	if(pathway == combo){
	  //TODO: how can we prevent duplicates?
	  is_unique_and_independent = false;
	  break;
	}
	is_unique_and_independent = ((combo & pathway) != pathway);
      }

      if(is_unique_and_independent){
	bins[batch_size * (i+1) + tid] = i;
	count++;
      } else {
	bins[batch_size * (i+1) + tid] = -1;
      }
    }

    //Assumes first item is 0 to start
    if(output_start == num_inputs)
      bins[tid] = 0;

    bins[tid] += count;
  }
}

void sortInputsOutputs(float *d_metaboliteCoefficients, int pathwayCounts, 
		       BinaryVector *d_reactions, int metaboliteCount, int numInputs, 
		       int numOutputs, int metaboliteToRemove){
  //call the kernel on inputs and outputs
  int numBlocks = (pathwayCounts / MAX_THREADS_PER_BLOCK ) + 1;
  sortInputPathways <<< numBlocks, MAX_THREADS_PER_BLOCK >>> 
    (d_reactions, d_metaboliteCoefficients, 
     0, pathwayCounts, pathwayCounts,
     metaboliteCount, metaboliteToRemove);

  numBlocks = ((pathwayCounts - numInputs) / MAX_THREADS_PER_BLOCK ) + 1;
  sortOutputPathways <<< numBlocks, MAX_THREADS_PER_BLOCK >>> 
    (d_reactions, d_metaboliteCoefficients, 
     numInputs, pathwayCounts-numInputs, pathwayCounts,
     metaboliteCount, metaboliteToRemove);

}

void dependencyCheck(int numInputs, int numOutputs, int batch_number){
  int numBlocks = (numInputs / MAX_THREADS_PER_BLOCK ) + 1;
  dependencyCheck <<< numBlocks , MAX_THREADS_PER_BLOCK >>> 
    (d_binaryVectors, d_combinationBins, batchSize, numInputs,
     numInputs + (batch_number * batchSize), //start of next batch of outputs
     numOutputs, //start of non-participating
     pathwayCount);
}
